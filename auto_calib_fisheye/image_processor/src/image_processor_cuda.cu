#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 - 2023, VINAI Artificial Intelligence Application and Research JSC.
 * All rights reserved. All information contained here is proprietary and confidential to VinAI.
 * Any use, reproduction, or disclosure without the written permission
 * of VinAI is prohibited.
 */
#include <hip/hip_runtime_api.h>

#include <fstream>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc.hpp>

#include "image_processor_cuda.h"

namespace perception {
namespace imgproc {
constexpr int FISHEYE_COLS     = 1280;
constexpr int FISHEYE_ROWS     = 800;
constexpr int FISHEYE_CHANNELS = 3;
constexpr int TOP_COLS         = 800;
constexpr int TOP_ROWS         = 900;
constexpr int TOP_CHANNELS     = 3;
constexpr int MAP_COLS         = TOP_COLS;
constexpr int MAP_ROWS         = TOP_ROWS;
constexpr int MAP_CHANNELS     = 2;
constexpr int MASK_COLS        = TOP_COLS;
constexpr int MASK_ROWS        = TOP_ROWS;
constexpr int MASK_CHANNELS    = 1;

constexpr int BLOCKS_PER_DIM_X    = 25;
constexpr int BLOCKS_PER_DIM_Y    = 29;
constexpr int THREADS_PER_BLOCK_X = 32;
constexpr int THREADS_PER_BLOCK_Y = 32;
constexpr int COLS_PER_THREAD     = 1;
constexpr int ROWS_PER_THREAD     = 1;

static __global__ void remapAndMaskImage(const unsigned char* __restrict fisheyePtr,
                                         const short* __restrict mapPtr,
                                         const float* __restrict maskPtr,
                                         float* __restrict midtopPtr)
{
    const int y0 = ROWS_PER_THREAD * (THREADS_PER_BLOCK_Y * blockIdx.y + threadIdx.y);
    int y1       = ROWS_PER_THREAD * (THREADS_PER_BLOCK_Y * blockIdx.y + threadIdx.y + 1);
    const int x0 = COLS_PER_THREAD * (THREADS_PER_BLOCK_X * blockIdx.x + threadIdx.x);
    int x1       = COLS_PER_THREAD * (THREADS_PER_BLOCK_X * blockIdx.x + threadIdx.x + 1);

    x1 = (x1 < TOP_COLS) ? x1 : TOP_COLS;
    y1 = (y1 < TOP_ROWS) ? y1 : TOP_ROWS;
    for (int y = y0; y < y1; y++)
    {
        for (int x = x0; x < x1; x++)
        {
            const int mapID    = MAP_CHANNELS * (y * MAP_COLS + x);
            const short coordX = mapPtr[mapID + 0];
            const short coordY = mapPtr[mapID + 1];
            if ((coordX < 0) || (coordX > FISHEYE_COLS - 1) || (coordY < 0) ||
                (coordY > FISHEYE_ROWS - 1))
            {
                continue;
            }

            const int fisheyeID  = FISHEYE_CHANNELS * (coordY * FISHEYE_COLS + coordX);
            const int topID      = TOP_CHANNELS * (y * TOP_COLS + x);
            const int maskID     = MASK_CHANNELS * (y * MASK_COLS + x);
            const auto intensR   = fisheyePtr[fisheyeID + 0];
            const auto intensG   = fisheyePtr[fisheyeID + 1];
            const auto intensB   = fisheyePtr[fisheyeID + 2];
            const auto alpha     = maskPtr[maskID];
            midtopPtr[topID + 0] = intensR * alpha;
            midtopPtr[topID + 1] = intensG * alpha;
            midtopPtr[topID + 2] = intensB * alpha;
        }
    }
}

static __global__ void combineImagePortion(const float* __restrict midtop0Ptr,
                                           const float* __restrict midtop1Ptr,
                                           const float* __restrict midtop2Ptr,
                                           const float* __restrict midtop3Ptr,
                                           unsigned char* __restrict topPtr)
{
    const int y0 = ROWS_PER_THREAD * (THREADS_PER_BLOCK_Y * blockIdx.y + threadIdx.y);
    int y1       = ROWS_PER_THREAD * (THREADS_PER_BLOCK_Y * blockIdx.y + threadIdx.y + 1);
    const int x0 = COLS_PER_THREAD * (THREADS_PER_BLOCK_X * blockIdx.x + threadIdx.x);
    int x1       = COLS_PER_THREAD * (THREADS_PER_BLOCK_X * blockIdx.x + threadIdx.x + 1);

    x1 = (x1 < TOP_COLS) ? x1 : TOP_COLS;
    y1 = (y1 < TOP_ROWS) ? y1 : TOP_ROWS;
    for (int y = y0; y < y1; y++)
    {
        for (int x = x0; x < x1; x++)
        {
            const int id = TOP_CHANNELS * (y * TOP_COLS + x);
            for (int z = 0; z < TOP_CHANNELS; z++)
            {
                topPtr[id + z] = midtop0Ptr[id + z] + midtop1Ptr[id + z] + midtop2Ptr[id + z] +
                                 midtop3Ptr[id + z];
            }
        }
    }
}

ImageProcessorCuda::ImageProcessorCuda(const ImageProcessorConfig& config) : IImageProcessor(config)
{
    calibDir = config.calib_dir();
}

ImageProcessorCuda::~ImageProcessorCuda()
{
    for (int id = 0; id < NUM_CAMS; id++)
    {
        hipFree(mapPtr[id]);
        hipFree(maskPtr[id]);
        hipHostFree(fisheyePtr[id]);
        hipFree(midtopPtr[id]);
    }
}

bool ImageProcessorCuda::init()
{
    for (int id = 0; id < NUM_CAMS; id++)
    {
        // Load and copy maps and masks to device memory
        mapPtr[id]  = loadAndCopyMap(calibDir + "topview_rgb/map" + std::to_string(id) + ".txt",
                                     MAP_COLS, MAP_ROWS);
        maskPtr[id] = loadAndCopyMask(calibDir + "topview_rgb/mask" + std::to_string(id) + ".png");

        // Allocate pinned memory for fisheye images
        fisheyePtr[id] = allocateFisheye();

        // Allocate device memory to mid-topviews
        midtopPtr[id] = allocateMidtopview(TOP_ROWS * TOP_COLS * TOP_CHANNELS);
    }

    return true;
}

bool ImageProcessorCuda::init(const UVLists& uvLists)
{
    for (int id = 0; id < NUM_CAMS; id++)
    {
        // Load and copy maps and masks to device memory
        auto& map         = uvLists[id];
        const int mapSize = map.size() * sizeof(short);
        hipMalloc((void**)&mapPtr[id], mapSize);
        hipMemcpy(mapPtr[id], map.data(), mapSize, hipMemcpyHostToDevice);

        maskPtr[id] = loadAndCopyMask(calibDir + "topview_rgb/mask" + std::to_string(id) + ".png");

        // Allocate pinned memory for fisheye images
        fisheyePtr[id] = allocateFisheye();

        // Allocate device memory to mid-topviews
        midtopPtr[id] = allocateMidtopview(TOP_ROWS * TOP_COLS * TOP_CHANNELS);
    }

    return true;
}

void ImageProcessorCuda::createTopViewImage(const cv::Mat& fisheye0, const cv::Mat& fisheye1,
                                            const cv::Mat& fisheye2, const cv::Mat& fisheye3,
                                            cv::Mat& topImg)
{
    // Allocate device memory to topview
    topImg = cv::Mat(TOP_ROWS, TOP_COLS, CV_8UC3);
    unsigned char* topPtr;
    const int topSize = topImg.total() * topImg.elemSize();
    hipMalloc((void**)&topPtr, topSize);
    GET_LAST_CUDA_ERRORS();

    // Perform topview generation: remapping & masking
    hipStream_t topStream[NUM_CAMS];
    for (int id = 0; id < NUM_CAMS; id++)
    {
        hipStreamCreate(&topStream[id]);
    }
    const cv::Mat fisheye[NUM_CAMS] = {fisheye0, fisheye1, fisheye2, fisheye3};
    for (int id = 0; id < NUM_CAMS; id++)
    {
        hipMemcpyAsync((void*)fisheyePtr[id], fisheye[id].data,
                        fisheye[id].total() * fisheye[id].elemSize(), hipMemcpyHostToDevice,
                        topStream[id]);
        remapAndMaskImage<<<dim3{BLOCKS_PER_DIM_X, BLOCKS_PER_DIM_Y, 1},
                            dim3{THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1}, 0, topStream[id]>>>(
            fisheyePtr[id], mapPtr[id], maskPtr[id], midtopPtr[id]);
    }
    hipDeviceSynchronize();
    for (int id = 0; id < NUM_CAMS; id++)
    {
        hipStreamDestroy(topStream[id]);
    }
    GET_LAST_CUDA_ERRORS();

    // Perform topview generation: combining
    combineImagePortion<<<dim3{BLOCKS_PER_DIM_X, BLOCKS_PER_DIM_Y, 1},
                          dim3{THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1}>>>(
        midtopPtr[0], midtopPtr[1], midtopPtr[2], midtopPtr[3], topPtr);
    hipMemcpy(topImg.data, topPtr, topImg.total() * topImg.elemSize(), hipMemcpyDeviceToHost);
    GET_LAST_CUDA_ERRORS();

    // Free resources
    hipFree(topPtr);
    GET_LAST_CUDA_ERRORS();
}

unsigned char* ImageProcessorCuda::allocateFisheye()
{
    unsigned char* fisheyePtr;
    constexpr int fisheyeSize = FISHEYE_COLS * FISHEYE_ROWS * FISHEYE_CHANNELS;
    hipHostMalloc((void**)&fisheyePtr, fisheyeSize);
    return fisheyePtr;
}
}  // namespace imgproc
}  // namespace perception